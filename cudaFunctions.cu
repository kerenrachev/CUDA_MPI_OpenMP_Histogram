#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudaFunctions.h"

__global__  void calculateHist(int *arr, int numElements, int* output) {
    int index = blockIdx.x *256 + threadIdx.x;
    __shared__ int hist[256];
    hist[index] = 0;
    
    if (index < numElements )
    {
        atomicAdd(&hist[arr[index]],1);
    }
    __syncthreads();

    // The thread with index 0 will merge the shared memory histogram with the global histogram using atomicAdd()
    if(threadIdx.x==0)
    {
        for(int i=0; i< 256; i++)
        {
            output[i] = 0;
            atomicAdd(&output[i],hist[i]);   
        }
    }
}
int* computeOnGPU(int *data, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t size = numElements * sizeof(float);
    size_t outputSize = 256 * sizeof(float);

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate memory on GPU to copy the output from the host
    int *d_out;
    err = hipMalloc((void **)&d_out, outputSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy result array histogram from host to the GPU memory
    err = hipMemcpy(d_out, data, outputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(float(numElements)/ float(threadsPerBlock));
    calculateHist<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements,d_out);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(data, d_A, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy local numbers array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int* output = (int*)malloc(sizeof(int)*256);
    for(int i=0 ; i<256; i++)
    {
        output[i]=0;
    }
    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(output, d_out, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy output array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_out) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return output;
}
